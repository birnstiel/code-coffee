/*
A simple CUDA C++ code to add 2 float arrays in different ways.
Stolen from https://devblogs.nvidia.com/even-easier-introduction-cuda/
K. Huber 2018
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <time.h>



// CUDA Kernel functions to add the elements of two arrays on the GPU

// one thread
__global__ void add_1(int n, float *x, float *y)
{
    for(int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

// multiple threads in 1 block
__global__ void add_t(int n, float *x, float *y)
{
    int index = threadIdx.x;
    int stride = blockDim.x;
    for(int i = index; i < n; i+= stride)
      y[i] = x[i] + y[i];
}

// multiple blocks and threads
__global__ void add_bt(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  time_t start_time, end_time;
  time(&start_time);
  int N = 1e7; // 10M elements

  // Allocate Unified Memory -- accessible from CPU or GPU
  float *x, *y;
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on the GPU
  //  add_1<<<1,1>>>(N,x,y);
  
//  add_t<<<1, 512>>>(N,x,y);

  int blockSize = 512;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add_bt<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);

  time(&end_time);
  std::cout << "time: " << difftime(end_time, start_time) << " s" << std::endl;

  return 0;
}

